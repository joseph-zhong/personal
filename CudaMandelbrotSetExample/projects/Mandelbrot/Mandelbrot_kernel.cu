#include "hip/hip_runtime.h"
#include <stdio.h>
#include "cutil.h"
#include "Mandelbrot_kernel.h"

// The dimensions of the thread block
#define BLOCKDIM_X 8
#define BLOCKDIM_Y 8

#define ABS(n) ((n) < 0 ? -(n) : (n))

// This function sets the DS number A equal to the double precision floating point number B. 
inline void dsdeq(float &a0, float &a1, double b)
{
    a0 = (float)b;
    a1 = (float)(b - a0);
} // dsdcp

// This function sets the DS number A equal to the single precision floating point number B. 
__device__ inline void dsfeq(float &a0, float &a1, float b)
{
    a0 = b;
    a1 = 0.0f;
} // dsfeq

// This function computes c = a + b.
__device__ inline void dsadd(float &c0, float &c1, const float a0, const float a1, const float b0, const float b1)
{
    // Compute dsa + dsb using Knuth's trick.
    float t1 = a0 + b0;
    float e = t1 - a0;
    float t2 = ((b0 - e) + (a0 - (t1 - e))) + a1 + b1;

    // The result is t1 + t2, after normalization.
    c0 = e = t1 + t2;
    c1 = t2 - (e - t1);
} // dsadd

// This function computes c = a - b.
__device__ inline void dssub(float &c0, float &c1, const float a0, const float a1, const float b0, const float b1)
{
    // Compute dsa - dsb using Knuth's trick.
    float t1 = a0 - b0;
    float e = t1 - a0;
    float t2 = ((-b0 - e) + (a0 - (t1 - e))) + a1 - b1;

    // The result is t1 + t2, after normalization.
    c0 = e = t1 + t2;
    c1 = t2 - (e - t1);
} // dssub

// This function multiplies DS numbers A and B to yield the DS product C.
__device__ inline void dsmul(float &c0, float &c1, const float a0, const float a1, const float b0, const float b1)
{
	// This splits dsa(1) and dsb(1) into high-order and low-order words.
	float cona = a0 * 8193.0f;
	float conb = b0 * 8193.0f;
	float sa1 = cona - (cona - a0);
	float sb1 = conb - (conb - b0);
	float sa2 = a0 - sa1;
	float sb2 = b0 - sb1;

	// Multilply a0 * b0 using Dekker's method.
	float c11 = a0 * b0;
	float c21 = (((sa1 * sb1 - c11) + sa1 * sb2) + sa2 * sb1) + sa2 * sb2;

    // Compute a0 * b1 + a1 * b0 (only high-order word is needed).
    float c2 = a0 * b1 + a1 * b0;

    // Compute (c11, c21) + c2 using Knuth's trick, also adding low-order product.
    float t1 = c11 + c2;
    float e = t1 - c11;
    float t2 = ((c2 - e) + (c11 - (t1 - e))) + c21 + a1 * b1;

    // The result is t1 + t2, after normalization.
    c0 = e = t1 + t2;
    c1 = t2 - (e - t1);
} // dsmul

// The core Mandelbrot CUDA GPU calculation function
#if 1
// Unrolled version
__device__ inline int CalcMandelbrot(const float xPos, const float yPos, const int crunch)
{
    float y = yPos;
    float x = xPos;
    float yy = y * y;
    float xx = x * x;
    int i = crunch;

    do {
		// Iteration 1
		if (xx + yy > 4.0f)
			return i - 1;
        y = x * y * 2.0f + yPos;
        x = xx - yy + xPos;
        yy = y * y;
        xx = x * x;

		// Iteration 2
		if (xx + yy > 4.0f)
			return i - 2;
        y = x * y * 2.0f + yPos;
        x = xx - yy + xPos;
        yy = y * y;
        xx = x * x;

		// Iteration 3
		if (xx + yy > 4.0f)
			return i - 3;
        y = x * y * 2.0f + yPos;
        x = xx - yy + xPos;
        yy = y * y;
        xx = x * x;

		// Iteration 4
		if (xx + yy > 4.0f)
			return i - 4;
        y = x * y * 2.0f + yPos;
        x = xx - yy + xPos;
        yy = y * y;
        xx = x * x;

		// Iteration 5
		if (xx + yy > 4.0f)
			return i - 5;
        y = x * y * 2.0f + yPos;
        x = xx - yy + xPos;
        yy = y * y;
        xx = x * x;

		// Iteration 6
		if (xx + yy > 4.0f)
			return i - 6;
        y = x * y * 2.0f + yPos;
        x = xx - yy + xPos;
        yy = y * y;
        xx = x * x;

		// Iteration 7
		if (xx + yy > 4.0f)
			return i - 7;
        y = x * y * 2.0f + yPos;
        x = xx - yy + xPos;
        yy = y * y;
        xx = x * x;

		// Iteration 8
		if (xx + yy > 4.0f)
			return i - 8;
        y = x * y * 2.0f + yPos;
        x = xx - yy + xPos;
        yy = y * y;
        xx = x * x;

		// Iteration 9
		if (xx + yy > 4.0f)
			return i - 9;
        y = x * y * 2.0f + yPos;
        x = xx - yy + xPos;
        yy = y * y;
        xx = x * x;

		// Iteration 10
		if (xx + yy > 4.0f)
			return i - 10;
        y = x * y * 2.0f + yPos;
        x = xx - yy + xPos;
        yy = y * y;
        xx = x * x;       

		// Iteration 11
		if (xx + yy > 4.0f)
			return i - 11;
        y = x * y * 2.0f + yPos;
        x = xx - yy + xPos;
        yy = y * y;
        xx = x * x;       

		// Iteration 12
		if (xx + yy > 4.0f)
			return i - 12;
        y = x * y * 2.0f + yPos;
        x = xx - yy + xPos;
        yy = y * y;
        xx = x * x;       

		// Iteration 13
		if (xx + yy > 4.0f)
			return i - 13;
        y = x * y * 2.0f + yPos;
        x = xx - yy + xPos;
        yy = y * y;
        xx = x * x;       

		// Iteration 14
		if (xx + yy > 4.0f)
			return i - 14;
        y = x * y * 2.0f + yPos;
        x = xx - yy + xPos;
        yy = y * y;
        xx = x * x;       

		// Iteration 15
		if (xx + yy > 4.0f)
			return i - 15;
        y = x * y * 2.0f + yPos;
        x = xx - yy + xPos;
        yy = y * y;
        xx = x * x;       

		// Iteration 16
		if (xx + yy > 4.0f)
			return i - 16;
        y = x * y * 2.0f + yPos;
        x = xx - yy + xPos;
        yy = y * y;
        xx = x * x;       

		// Iteration 17
		if (xx + yy > 4.0f)
			return i - 17;
        y = x * y * 2.0f + yPos;
        x = xx - yy + xPos;
        yy = y * y;
        xx = x * x;       

		// Iteration 18
		if (xx + yy > 4.0f)
			return i - 18;
        y = x * y * 2.0f + yPos;
        x = xx - yy + xPos;
        yy = y * y;
        xx = x * x;       

		// Iteration 19
		if (xx + yy > 4.0f)
			return i - 19;
        y = x * y * 2.0f + yPos;
        x = xx - yy + xPos;
        yy = y * y;
        xx = x * x;       

		// Iteration 20
        i -= 20;
		if ((i <= 0) || (xx + yy > 4.0f))
			return i;
        y = x * y * 2.0f + yPos;
        x = xx - yy + xPos;
        yy = y * y;
        xx = x * x;       
    } while (1);
} // CalcMandelbrot
#else
__device__ inline int CalcMandelbrot(const float xPos, const float yPos, const int crunch)
{
    float y = yPos;
    float x = xPos;
    float yy = y * y;
    float xx = x * x;
    int i = crunch;

    while (--i && (xx + yy < 4.0f)) {
        y = x * y * 2.0f + yPos;
        x = xx - yy + xPos;
        yy = y * y;
        xx = x * x;
    }
    return i // i > 0 ? crunch - i : 0;
} // CalcMandelbrot
#endif

// The core Mandelbrot calculation function in double precision
__device__ inline int CalcMandelbrotDS(const float xPos0, const float xPos1, const float yPos0, const float yPos1, const int crunch)
{
    float xx0, xx1;
    float yy0, yy1;
    float sum0, sum1;
    int i = crunch;

	float y0 = yPos0;	// y = yPos;
	float y1 = yPos1;
	float x0 = xPos0;	// x = xPos;
	float x1 = xPos1;
    dsmul(yy0, yy1, y0, y1, y0, y1);    // yy = y * y;
    dsmul(xx0, xx1, x0, x1, x0, x1);	// xx = x * x;
    dsadd(sum0, sum1, xx0, xx1, yy0, yy1);	// sum = xx + yy;
    while (--i && (sum0 + sum1 < 4.0f)) {
        dsmul(y0, y1, x0, x1, y0, y1);		// y = x * y * 2.0f + yPos;
        dsadd(y0, y1, y0, y1, y0, y1);
        dsadd(y0, y1, y0, y1, yPos0, yPos1);

        dssub(x0, x1, xx0, xx1, yy0, yy1);	//  x = xx - yy + xPos;
        dsadd(x0, x1, x0, x1, xPos0, xPos1);

		dsmul(yy0, yy1, y0, y1, y0, y1);    // yy = y * y;
		dsmul(xx0, xx1, x0, x1, x0, x1);	// xx = x * x;
		dsadd(sum0, sum1, xx0, xx1, yy0, yy1);	// sum = xx + yy;
    }
    return i;
} // CalcMandelbrotDS

// The Mandelbrot CUDA GPU thread function
__global__ void Mandelbrot0(uchar4 *dst, const int imageW, const int imageH, const int crunch, const float xOff, const float yOff, const float scale, const uchar4 colors, const int frame, const int animationFrame)
{
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if ((ix < imageW) && (iy < imageH)) {
		// Calculate the location
		const float xPos = (float)ix * scale + xOff;
		const float yPos = (float)iy * scale + yOff;
		      
        // Calculate the Mandelbrot index for the current location
        int m = CalcMandelbrot(xPos, yPos, crunch);
        m = m > 0 ? crunch - m : 0;
			
        // Convert the Madelbrot index into a color
        uchar4 color;
        if (m) {
			m += animationFrame;
			color.x = m * colors.x;
			color.y = m * colors.y;
			color.z = m * colors.z;
		} else {
			color.x = 0;
			color.y = 0;
			color.z = 0;
		}
		
        // Output the pixel
 		int pixel = imageW * iy + ix;
        if (frame == 0) {
			color.w = 0;
			dst[pixel] = color;
        } else {
			int frame1 = frame + 1;
			int frame2 = frame1 / 2;
			dst[pixel].x = (dst[pixel].x * frame + color.x + frame2) / frame1;
			dst[pixel].y = (dst[pixel].y * frame + color.y + frame2) / frame1;
			dst[pixel].z = (dst[pixel].z * frame + color.z + frame2) / frame1;
        }
    }
} // Mandelbrot0

// The Mandelbrot CUDA GPU thread function
__global__ void MandelbrotDS0(uchar4 *dst, const int imageW, const int imageH, const int crunch, const float xOff0, const float xOff1, const float yOff0, const float yOff1, const float scale, const uchar4 colors, const int frame, const int animationFrame)
{
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if ((ix < imageW) && (iy < imageH)) {
		// Calculate the location
		float xPos0 = (float)ix * scale;
		float xPos1 = 0.0f;
		float yPos0 = (float)iy * scale;
		float yPos1 = 0.0f;
		dsadd(xPos0, xPos1, xPos0, xPos1, xOff0, xOff1);
		dsadd(yPos0, yPos1, yPos0, yPos1, yOff0, yOff1);

        // Calculate the Mandelbrot index for the current location
        int m = CalcMandelbrotDS(xPos0, xPos1, yPos0, yPos1, crunch);
        m = m > 0 ? crunch - m : 0;
			
        // Convert the Madelbrot index into a color
        uchar4 color;
        if (m) {
			m += animationFrame;
			color.x = m * colors.x;
			color.y = m * colors.y;
			color.z = m * colors.z;
		} else {
			color.x = 0;
			color.y = 0;
			color.z = 0;
		}
		
        // Output the pixel
 		int pixel = imageW * iy + ix;
        if (frame == 0) {
			color.w = 0;
			dst[pixel] = color;
        } else {
			int frame1 = frame + 1;
			int frame2 = frame1 / 2;
			dst[pixel].x = (dst[pixel].x * frame + color.x + frame2) / frame1;
			dst[pixel].y = (dst[pixel].y * frame + color.y + frame2) / frame1;
			dst[pixel].z = (dst[pixel].z * frame + color.z + frame2) / frame1;
        }
    }
} // MandelbrotDS0

// Determine if two pixel colors are within tolerance
__device__ inline int CheckColors(const uchar4 &color0, const uchar4 &color1)
{
	int x = color1.x - color0.x;
	int y = color1.y - color0.y;
	int z = color1.z - color0.z;
	return (ABS(x) > 10) || (ABS(y) > 10) || (ABS(z) > 10);
} // CheckColors

// The Mandelbrot secondary AA pass CUDA GPU thread function
__global__ void Mandelbrot1(uchar4 *dst, const int imageW, const int imageH, const int crunch, const float xOff, const float yOff, const float scale, const uchar4 colors, const int frame, const int animationFrame)
{
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if ((ix < imageW) && (iy < imageH)) {
		// Get the current pixel color
 		int pixel = imageW * iy + ix;
		uchar4 pixelColor = dst[pixel];
		int count = 0;
		
		// Search for pixels out of tolerance surrounding the current pixel
		if (ix > 0)
			count += CheckColors(pixelColor, dst[pixel - 1]);
		if (ix + 1 < imageW)
			count += CheckColors(pixelColor, dst[pixel + 1]);
		if (iy > 0)
			count += CheckColors(pixelColor, dst[pixel - imageW]);
		if (iy + 1 < imageH)
			count += CheckColors(pixelColor, dst[pixel + imageW]);
		if (count) {
			// Calculate the location
			const float xPos = (float)ix * scale + xOff;
			const float yPos = (float)iy * scale + yOff;
			      
			// Calculate the Mandelbrot index for the current location
			int m = CalcMandelbrot(xPos, yPos, crunch);
			m = m > 0 ? crunch - m : 0;
	        
			// Convert the Madelbrot index into a color
			uchar4 color;
			if (m) {
				m += animationFrame;
				color.x = m * colors.x;
				color.y = m * colors.y;
				color.z = m * colors.z;
			} else {
				color.x = 0;
				color.y = 0;
				color.z = 0;
			}
			
			// Output the pixel
			int frame1 = frame + 1;
			int frame2 = frame1 / 2;
			dst[pixel].x = (pixelColor.x * frame + color.x + frame2) / frame1;
			dst[pixel].y = (pixelColor.y * frame + color.y + frame2) / frame1;
			dst[pixel].z = (pixelColor.z * frame + color.z + frame2) / frame1;
		}
    }
} // Mandelbrot1

// The Mandelbrot secondary AA pass CUDA GPU thread function
__global__ void MandelbrotDS1(uchar4 *dst, const int imageW, const int imageH, const int crunch, const float xOff0, const float xOff1, const float yOff0, const float yOff1, const float scale, const uchar4 colors, const int frame, const int animationFrame)
{
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if ((ix < imageW) && (iy < imageH)) {
		// Get the current pixel color
 		int pixel = imageW * iy + ix;
		uchar4 pixelColor = dst[pixel];
		int count = 0;
		
		// Search for pixels out of tolerance surrounding the current pixel
		if (ix > 0)
			count += CheckColors(pixelColor, dst[pixel - 1]);
		if (ix + 1 < imageW)
			count += CheckColors(pixelColor, dst[pixel + 1]);
		if (iy > 0)
			count += CheckColors(pixelColor, dst[pixel - imageW]);
		if (iy + 1 < imageH)
			count += CheckColors(pixelColor, dst[pixel + imageW]);
		if (count) {
			// Calculate the location
			float xPos0 = (float)ix * scale;
			float xPos1 = 0.0f;
			float yPos0 = (float)iy * scale;
			float yPos1 = 0.0f;
			dsadd(xPos0, xPos1, xPos0, xPos1, xOff0, xOff1);
			dsadd(yPos0, yPos1, yPos0, yPos1, yOff0, yOff1);
			      
			// Calculate the Mandelbrot index for the current location
			int m = CalcMandelbrotDS(xPos0, xPos1, yPos0, yPos1, crunch);
			m = m > 0 ? crunch - m : 0;
	        
			// Convert the Madelbrot index into a color
			uchar4 color;
			if (m) {
				m += animationFrame;
				color.x = m * colors.x;
				color.y = m * colors.y;
				color.z = m * colors.z;
			} else {
				color.x = 0;
				color.y = 0;
				color.z = 0;
			}
			
			// Output the pixel
			int frame1 = frame + 1;
			int frame2 = frame1 / 2;
			dst[pixel].x = (pixelColor.x * frame + color.x + frame2) / frame1;
			dst[pixel].y = (pixelColor.y * frame + color.y + frame2) / frame1;
			dst[pixel].z = (pixelColor.z * frame + color.z + frame2) / frame1;
		}
    }
} // MandelbrotDS1

// Increase the grid size by 1 if the image width or height does not divide evenly
// by the thread block dimensions
inline int iDivUp(int a, int b)
{
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
} // iDivUp

// The host CPU Mandebrot thread spawner
void RunMandelbrot0(uchar4 *dst, const int imageW, const int imageH, const int crunch, const double xOff, const double yOff, const double scale, const uchar4 colors, const int frame, const int animationFrame)
{
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));

	if (scale < 0.0000002f) {
		float x0, x1, y0, y1;
		dsdeq(x0, x1, xOff);
		dsdeq(y0, y1, yOff);
		MandelbrotDS0<<<grid, threads>>>(dst, imageW, imageH, crunch, x0, x1, y0, y1, (float)scale, colors, frame, animationFrame);
	} else
		Mandelbrot0<<<grid, threads>>>(dst, imageW, imageH, crunch, (float)xOff, (float)yOff, (float)scale, colors, frame, animationFrame);
    CUT_CHECK_ERROR("Mandelbrot kernel execution failed.\n");
} // RunMandelbrot0

// The host CPU Mandebrot thread spawner
void RunMandelbrot1(uchar4 *dst, const int imageW, const int imageH, const int crunch, const double xOff, const double yOff, const double scale, const uchar4 colors, const int frame, const int animationFrame)
{
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));

	if (scale < 0.0000002f) {
		float x0, x1, y0, y1;
		dsdeq(x0, x1, xOff);
		dsdeq(y0, y1, yOff);
		MandelbrotDS1<<<grid, threads>>>(dst, imageW, imageH, crunch, x0, x1, y0, y1, (float)scale, colors, frame, animationFrame);
	} else
		Mandelbrot1<<<grid, threads>>>(dst, imageW, imageH, crunch, (float)xOff, (float)yOff, (float)scale, colors, frame, animationFrame);
    CUT_CHECK_ERROR("Mandelbrot kernel execution failed.\n");
} // RunMandelbrot1
