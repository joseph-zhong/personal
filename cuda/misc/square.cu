// square.c


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void square(float *d_out, float *d_in) {
  int idx = threadIdx.x;
  float f = d_in[idx];
  d_out[idx] = f * f;  
}

int main(int argc, char **argv) {
  const int ARRAY_SIZE = 1024;
  const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

  // generate input array on host 
  float h_in[ARRAY_SIZE];
  for (int i = 0; i < ARRAY_SIZE; i++) {
    h_in[i] = float(i);
  }
  float h_out[ARRAY_SIZE];

  // declare GPU memory pointers
  float *d_in;
  float *d_out;

  // allocate GPU memory
  hipMalloc((void **) &d_in, ARRAY_BYTES);
  hipMalloc((void **) &d_out, ARRAY_BYTES);

  // transfer array to GPU
  hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

  // launch the kernel
  square<<<1, ARRAY_SIZE>>>(d_out, d_in);

  // syncronize the results back to CPU
  hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

  for (int i = 0; i < ARRAY_SIZE; i++) {
    printf("%f", h_out[i]);
    printf(((i % 4) != 3) ? "\t" : "\n");
  }

  // free GPU memory allocation
  hipFree(d_in);
  hipFree(d_out);

  return 0;
}
